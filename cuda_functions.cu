#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "definitions.h"
#include "cuda_functions.h"

__global__ void compute_picture_kernal(int* d_objects_matrix, int* d_picture_matrix, int object_id, int picture_dim, int object_dim, double matching_value, int* d_result_array, int work_area_dim);
__device__ double is_match(int* d_objects_matrix, int* d_picture_matrix, int picture_row_index, int picture_column_index, int picture_dim, int object_dim ,double matching_value);
void check_cuda_error(hipError_t err);

// This function copies the object to the GPU using the multi stream concept (private stream for each call [thread])
void copy_object_to_GPU(object_struct* object_struct){

  hipStream_t thread_cuda_stream;
  hipError_t err = hipSuccess;

  int block_dim = object_struct->object_dim;
  int object_size = pow(block_dim, 2);

  err = hipStreamCreate(&thread_cuda_stream);
  check_cuda_error(err);

  // Allocating memory for the object matrinx in the GPU
  err = hipMallocAsync(&object_struct->d_objects_matrix, sizeof(int) * object_size, thread_cuda_stream);
  check_cuda_error(err);
  
  // Copying the object matrix to the GPU
  err = hipMemcpyAsync(object_struct->d_objects_matrix, object_struct->object_matrix, sizeof(int) * object_size, hipMemcpyHostToDevice, thread_cuda_stream);
  check_cuda_error(err);

  // wating to the previous calls in the stream to be done
  err = hipStreamSynchronize(thread_cuda_stream);
  check_cuda_error(err);

  err = hipStreamDestroy(thread_cuda_stream); 
  check_cuda_error(err);
}


// This function called by each thread in the grid and each thread calculates if the object exists in the position
// according to the matching algorithm that given in the project file 
__device__ double is_match(int* d_objects_matrix, int* d_picture_matrix, int picture_row_index, int picture_column_index, int picture_dim, int object_dim ,double matching_value){
	
  int object_row_index , object_column_index;
  double matching_result = 0, curr_picture_cell, curr_object_cell;
  int curr_object_matrix_size = pow(object_dim, 2);


  // looping through the rows and columns to callculate if threre is a match according to the algorithm that given in the project file 
  for( object_row_index = 0; object_row_index < object_dim; object_row_index++){
    for( object_column_index = 0; object_column_index < object_dim; object_column_index++){

        curr_picture_cell = d_picture_matrix[picture_dim * (picture_row_index + object_row_index) + (picture_column_index + object_column_index)];

        curr_object_cell = d_objects_matrix[object_dim * object_row_index + object_column_index];

        matching_result += fabs((double)(((curr_picture_cell - curr_object_cell) / curr_picture_cell) / curr_object_matrix_size));  

        if(matching_result >= matching_value){

          return NOT_MATCH;
        }
      }
  }

  return matching_result;
}


// This function calls the function is_match() by a thread according to his position in the grid and the block
// for each possible position in the picture then it checks if there is a match according to the return value from is_match()
// and if true it fills the oject information and position in the result array
__global__ void compute_picture_kernal(int* d_objects_matrix, int* d_picture_matrix, int object_id, int picture_dim, int object_dim, double matching_value, int* d_result_array, int work_area_dim) {

  int picture_row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int picture_column_index = blockIdx.x * blockDim.x + threadIdx.x;

  if(picture_row_index < work_area_dim && picture_column_index <  work_area_dim){

    double result = is_match(d_objects_matrix, d_picture_matrix, picture_row_index, picture_column_index, picture_dim, object_dim, matching_value);

    if(result != NOT_MATCH){

      d_result_array[(picture_row_index * work_area_dim + picture_column_index) * FOUND_OBJECT_INFO_SIZE] = object_id;
      d_result_array[(picture_row_index * work_area_dim + picture_column_index) * FOUND_OBJECT_INFO_SIZE +1] = picture_row_index;
      d_result_array[(picture_row_index * work_area_dim + picture_column_index) * FOUND_OBJECT_INFO_SIZE + 2] = picture_column_index;
    }
  }
}


// This function checks for each possible/valid position in the picture if the curr object exists in it
// and it works in the multi stream concept (eeach thread that calls the function will create a private thread for itself)
int compute_picture_on_GPU(picture_struct* curr_picture_struct,  object_struct* objects_structs_array, double matching_value, int picture_dim, int objects_num){

  hipStream_t thread_cuda_stream;
  hipError_t err = hipSuccess;
  int work_area_dim, result_array_size;
  int picture_size = pow(picture_dim, 2);
  int grid_dim;
  int* h_result_array = NULL;
  int* d_result_array = NULL;
  // The block size is static which is the max warp size = 32 so the blick size is 32*32
  dim3 block_dim3 = dim3(WARP_SIZE, WARP_SIZE);

  err = hipStreamCreate(&thread_cuda_stream);
  check_cuda_error(err);

  // Allocating memory on the GPU for the picture matrix
  err = hipMallocAsync(&curr_picture_struct->d_picture_matrix, picture_size * sizeof(int), thread_cuda_stream);
  check_cuda_error(err);

  // Copying the picture matrix to the GPu
  err = hipMemcpyAsync(curr_picture_struct->d_picture_matrix,  curr_picture_struct->picture_matrix, picture_size * sizeof(int), hipMemcpyHostToDevice, thread_cuda_stream);
  check_cuda_error(err);
  
  // Looping through the objects and for each object we check in parallel if it exists in all the posible positions in the picture
  for (int object_index = 0; object_index < objects_num; object_index++) {

    // the working area dimintion is picture_dim - curr_object.dim + 1 (so the object calculation dosent get out of the picture dimintions)
    work_area_dim = picture_dim - objects_structs_array[object_index].object_dim + 1;
    result_array_size = work_area_dim * work_area_dim * FOUND_OBJECT_INFO_SIZE;

    // Allocating memory on the host for the existing checks of the object in all the posible positions
    h_result_array = (int*) malloc(result_array_size * sizeof(int));
    if(h_result_array == NULL){ fprintf(stderr, "Host Failed To Malloc\n"); exit(EXIT_FAILURE); }

    // Allocating memory on the GPU for the existing checks of the object in all the posible positions
    err = hipMallocAsync(&d_result_array, result_array_size * sizeof(int), thread_cuda_stream);
    check_cuda_error(err);
    
    // Initializing the result array with -1 so we can know if the object exists in the position or not
    err = hipMemsetAsync(d_result_array, -1, result_array_size * sizeof(int), thread_cuda_stream);
    check_cuda_error(err);

    // Calculating the grid dimintion according to statis block dim which is the max warp size = 32 so the block size is 32*32
    grid_dim = work_area_dim % WARP_SIZE == 0 ? (work_area_dim / WARP_SIZE) : (work_area_dim / WARP_SIZE) + 1;

    dim3 grid_dim3 = dim3(grid_dim, grid_dim);

    // Calling the kernal to ckeck the exist of the object in all the posibble positions (using the thread private stream)
    compute_picture_kernal<<<grid_dim3, block_dim3, 0, thread_cuda_stream>>>(objects_structs_array[object_index].d_objects_matrix, curr_picture_struct->d_picture_matrix, objects_structs_array[object_index].object_id, picture_dim, objects_structs_array[object_index].object_dim, matching_value, d_result_array, work_area_dim);
    
    // copying the results array to the host
    err = hipMemcpyAsync(h_result_array, d_result_array, result_array_size * sizeof(int), hipMemcpyDeviceToHost, thread_cuda_stream);
    check_cuda_error(err);

    // wating till all the work on the current object have been done
    err = hipStreamSynchronize(thread_cuda_stream);
    check_cuda_error(err);
    
    // updating the found objects info for the picture by calling the function update_found_objects_info()
    update_found_objects_info(curr_picture_struct, h_result_array, work_area_dim);
      
    free(h_result_array);
    
    err = hipFreeAsync(d_result_array, thread_cuda_stream);
    check_cuda_error(err);

    // if three objects where foiund breake the loop
    if (curr_picture_struct->found_objects_num == TO_FIND_OBJECTS_NUM){ break; }   
  }

  err = hipFreeAsync(curr_picture_struct->d_picture_matrix, thread_cuda_stream);
  check_cuda_error(err);

  // wating till the free finishes sueccesfully to destroy the stream after that
  err = hipStreamSynchronize(thread_cuda_stream);
  check_cuda_error(err);

  err = hipStreamDestroy(thread_cuda_stream);
  check_cuda_error(err);

  return 0;
}

// This function updates the found objects information in the picture struct after the the computation of that object
// have been done on the GPU h_result_array contains the object information fo each possible position in the picture 
void update_found_objects_info(picture_struct* curr_picture_struct, int* h_result_array, int work_area_dim){

  int result_objects_index = curr_picture_struct->found_objects_num;

  for(int i = 0; i < work_area_dim * work_area_dim; i++){

    if (h_result_array[i * FOUND_OBJECT_INFO_SIZE] != -1){

      curr_picture_struct->found_objects_info[result_objects_index * FOUND_OBJECT_INFO_SIZE] = h_result_array[i * FOUND_OBJECT_INFO_SIZE];
      curr_picture_struct->found_objects_info[result_objects_index * FOUND_OBJECT_INFO_SIZE + 1] = h_result_array[i * FOUND_OBJECT_INFO_SIZE + 1];
      curr_picture_struct->found_objects_info[result_objects_index * FOUND_OBJECT_INFO_SIZE + 2] = h_result_array[i * FOUND_OBJECT_INFO_SIZE + 2];

      curr_picture_struct->found_objects_num += 1;

      break;      
    }
  }
}

// This function frees the manually allocated memories for the objects on the GPU 
// using the multi streams concept
void free_objects_on_GPU(object_struct* objects_structs_array,  int objects_num){

  int object_index;
  hipStream_t free_streams[objects_num];
  hipError_t err = hipSuccess;

  for (object_index = 0; object_index < objects_num; object_index++) {

     err = hipStreamCreate(&free_streams[object_index]);
    check_cuda_error(err);

    err = hipFreeAsync(objects_structs_array[object_index].d_objects_matrix, free_streams[object_index]);
    check_cuda_error(err);
  }

  // wating for all the works (frees) on all the streams to be done
  err = hipDeviceSynchronize();
  check_cuda_error(err);

  for (object_index = 0; object_index < objects_num; object_index++) {

    err = hipStreamDestroy(free_streams[object_index]); 
    check_cuda_error(err);
  }
}

// This function checks if threre is a cuda error if true it prints the cuda error string and exits/finishes the program
void check_cuda_error(hipError_t err){

  if(err == hipSuccess){
    return;
  }

  fprintf(stderr, "Cuda Error: %s)!\n", hipGetErrorString(err)); 
  exit(EXIT_FAILURE); 
}
